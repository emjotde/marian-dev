#include <hip/hip_runtime.h>
#include <iostream>

#include "tensors/device.h"
#include "tensors/gpu/cuda_helpers.h"

namespace marian {
namespace gpu {

Device::~Device() {
  hipSetDevice(deviceId_.no);
  if(data_) {
    CUDA_CHECK(hipFree(data_));
  }
  hipDeviceSynchronize();
}

void Device::reserve(size_t size) {
  size = align(size);
  hipSetDevice(deviceId_.no);

  ABORT_IF(size < size_ || size == 0,
           "New size must be larger than old size and larger than 0");

  if(data_) {
    // Allocate memory by going through host memory
    uint8_t *temp = new uint8_t[size_];
    CUDA_CHECK(hipMemcpy(temp, data_, size_, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(data_));
    CUDA_CHECK(hipMalloc(&data_, size));
    CUDA_CHECK(hipMemcpy(data_, temp, size_, hipMemcpyHostToDevice));
    delete[] temp;
  } else {
    CUDA_CHECK(hipMalloc(&data_, size));
  }

  size_ = size;
}
}  // namespace gpu
}  // namespace marian
