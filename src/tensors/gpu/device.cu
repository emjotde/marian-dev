#include <hip/hip_runtime.h>
#include <iostream>

#include "tensors/device.h"
#include "tensors/gpu/cuda_helpers.h"

namespace marian {
namespace gpu {

Device::~Device() {
  hipSetDevice(deviceId_.no);
  if(data_) {
    CUDA_CHECK(hipFree(data_));
  }
  hipDeviceSynchronize();
}

void Device::reserve(size_t size) {
  size = align(size);
  hipSetDevice(deviceId_.no);

  ABORT_IF(size < size_ || size == 0,
           "New size must be larger than old size and larger than 0");

  if(data_) {
    // Allocate memory by going through host memory
    uint8_t *temp = new uint8_t[size_]; // TODO: use std::vector
    CUDA_CHECK(hipMemcpy(temp, data_, size_, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(data_));
    LOG(info, "re-allocating {} bytes on device {} (with CPU detour)", size, deviceId_.no);
    CUDA_CHECK(hipMalloc(&data_, size));
    CUDA_CHECK(hipMemcpy(data_, temp, size_, hipMemcpyHostToDevice));
    delete[] temp;
  } else {
    LOG(info, "re-allocating {} bytes in device {} (without CPU detour)", size, deviceId_.no);
    CUDA_CHECK(hipMalloc(&data_, size));
  }

  size_ = size;
}
}  // namespace gpu
}  // namespace marian
