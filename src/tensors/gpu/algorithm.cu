#include "hip/hip_runtime.h"
#include "tensors/gpu/algorithm.h"

#include "kernels/cuda_helpers.h"

namespace marian {
  namespace gpu {
    void copy(Ptr<Backend> backend, const float* begin, const float* end, float* dest) {
      CUDA_CHECK(hipSetDevice(backend->getDevice().no));
      CudaCopy(begin, end, dest);
      CUDA_CHECK(hipStreamSynchronize(0));
    }

    __global__ void gFill(float *d_in, int size, float val) {
      for(int bid = 0; bid < size; bid += blockDim.x * gridDim.x) {
        int index = bid + threadIdx.x + blockDim.x * blockIdx.x;
        if(index < size) {
          d_in[index] = val;
        }
      }
    }

    void fill(Ptr<Backend> backend, float* begin, float* end, float value) {
      CUDA_CHECK(hipSetDevice(backend->getDevice().no));
      int size = end - begin;
      int threads = std::min(512, size);
      int blocks = (size / threads) + (size % threads != 0);
      gFill<<<blocks, threads>>>(begin, size, value);
      CUDA_CHECK(hipStreamSynchronize(0));
    }
  }
}
