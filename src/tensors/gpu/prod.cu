#include "hip/hip_runtime.h"

#include <hipblas.h>

// clang-format off
#include "tensors/gpu/prod.h"
#include "tensors/gpu/backend.h"
#include "tensors/gpu/cuda_helpers.h"
// clang-format on

namespace marian {

namespace gpu {

void Prod(marian::Tensor C,
          marian::Tensor A,
          marian::Tensor B,
          bool transA,
          bool transB,
          float beta,
          float scalar) {
  hipSetDevice(C->getDevice().no);
  float alpha = scalar;

  size_t m = A->shape().elements() / A->shape().back();
  size_t k = A->shape().back();
  if(transA)
    std::swap(m, k);

  size_t l = B->shape().elements() / B->shape().back();
  size_t n = B->shape().back();
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape().back();
  size_t ldb = B->shape().back();
  size_t ldc = B->shape().back();

  if(transB)
    ldc = B->shape().elements() / B->shape().back();

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  auto cublasHandle = std::static_pointer_cast<gpu::Backend>(C->getBackend())
                          ->getCublasHandle();

#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
#endif

  hipblasSgemm(cublasHandle,
              opB,
              opA,
              n,
              m,
              k,
              &alpha,
              B->data(),
              ldb,
              A->data(),
              lda,
              &beta,
              C->data(),
              ldc);
#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH);
#endif
}

void ProdWithBias(marian::Tensor C,
          const marian::Tensor A,
          const marian::Tensor B,
          const marian::Tensor bias,
          bool transA,
          bool transB,
          float beta,
          float scalar) {
  marian::gpu::Prod(C, A, B, transA, transB, beta, scalar);
  marian::gpu::Add(functional::_1, 1.f, C, bias);
}


void ProdBatched(marian::Tensor C,
                 const marian::Tensor A,
                 const marian::Tensor B,
                 bool transA,
                 bool transB,
                 float beta,
                 float scalar) {
  hipSetDevice(C->getDevice().no);
  float alpha = scalar;

  size_t batchA = A->shape().elements() / (A->shape()[-1] * A->shape()[-2]);
  size_t batchB = B->shape().elements() / (B->shape()[-1] * B->shape()[-2]);

  size_t m = A->shape()[-2];
  size_t k = A->shape()[-1];
  if(transA)
    std::swap(m, k);

  size_t l = B->shape()[-2];
  size_t n = B->shape()[-1];
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape()[-1];
  size_t ldb = B->shape()[-1];
  size_t ldc = B->shape()[-1];

  if(transB)
    ldc = B->shape()[-2];

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  auto cublasHandle = std::static_pointer_cast<gpu::Backend>(C->getBackend())
                          ->getCublasHandle();

#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
#endif
  hipblasSgemmStridedBatched(cublasHandle,
                            opB,
                            opA,
                            n,
                            m,
                            k,
                            &alpha,
                            B->data(),
                            ldb,
                            batchB == 1 ? 0 : n * k,
                            A->data(),
                            lda,
                            batchA == 1 ? 0 : m * k,
                            &beta,
                            C->data(),
                            ldc,
                            n * m,
                            std::max(batchA, batchB));
#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH);
#endif
}
}
}
