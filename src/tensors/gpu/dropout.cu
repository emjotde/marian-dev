#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>

// clang-format off
#include "tensors/tensor_operators.h"
#include "tensors/gpu/backend.h"
// clang-format on

#define CUDA_CALL(x)                                  \
  do {                                                \
    if((x) != hipSuccess) {                          \
      printf("Error at %s:%d\n", __FILE__, __LINE__); \
      exit(1);                                        \
    }                                                 \
  } while(0)

#define CURAND_CALL(x)                                \
  do {                                                \
    if((x) != HIPRAND_STATUS_SUCCESS) {                \
      printf("Error at %s:%d\n", __FILE__, __LINE__); \
      exit(1);                                        \
    }                                                 \
  } while(0)

namespace marian {
namespace gpu {

__global__ void gScale(float* data, int n, float p) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  while(index < n) {
    data[index] = (data[index] < p) / p;
    index += gridDim.x * blockDim.x;
  }
}

void Dropout(Tensor tensor, float p) {
  auto gpuBackend
      = std::static_pointer_cast<gpu::Backend>(tensor->getBackend());
  hiprandGenerator_t gen = gpuBackend->getCurandGenerator();
  int n = tensor->size();
  CURAND_CALL(hiprandGenerateUniform(gen, tensor->data(), n));

  int numThreads = std::min(n, 512);
  int numBlocks = n / numThreads + (n % numThreads != 0);

  gScale<<<numBlocks, numThreads>>>(tensor->data(), n, 1.f - p);
}
}  // namespace gpu
}  // namespace marian
