#include "kernels/cudnn_wrappers.h"

#ifdef CUDNN

#include <hipDNN.h>

#define CUDNN_CALL(x)                 \
  do {                                \
    if((x) != HIPDNN_STATUS_SUCCESS) { \
      printf("Error (%s) at %s:%d\n", \
             hipdnnGetErrorString(x),  \
             __FILE__,                \
             __LINE__);               \
    }                                 \
  } while(0)

namespace marian {

CUDNNWrapper::CUDNNWrapper() {
  CUDNN_CALL(hipdnnCreate(&cudnnHandle_));
}

CUDNNWrapper::~CUDNNWrapper() {
  hipdnnDestroy(cudnnHandle_);
}

void CUDNNWrapper::setCudnnTensor(hipdnnTensorDescriptor_t& desc, Tensor x) {
  setCudnnTensor(desc, x->shape());
}

void CUDNNWrapper::setCudnnTensor(
    hipdnnTensorDescriptor_t& desc,
    const Shape& shape) {
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        desc,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        shape[0],
        shape[1],
        shape[2],
        shape[3]));
}

ConvolutionWrapper::ConvolutionWrapper(
    const Shape& kernelShape,
    const Shape& biasShape,
    int hPad,
    int wPad,
    int hStride,
    int wStride) {
  setKernelDescriptor(kernelShape);
  setConvDescriptor(hPad, wPad, hStride, wStride);
  setCudnnTensor(biasDesc_, biasShape);
}

void ConvolutionWrapper::getOutputShape(
    const Shape& xShape,
    Shape& shape) {
  hipdnnTensorDescriptor_t xDesc;
  setCudnnTensor(xDesc, xShape);
  shape.resize(4);
  CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        convDesc_,
        xDesc,
        kernelDesc_,
        shape.data(),
        shape.data() + 1,
        shape.data() + 2,
        shape.data() + 3));
  hipdnnDestroyTensorDescriptor(xDesc);
}

void ConvolutionWrapper::forward(
    Tensor x,
    Tensor kernels,
    Tensor bias,
    Tensor y) {
  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, x);
  setCudnnTensor(yDesc, y);

  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipSetDevice(x->getDevice());

  CUDNN_CALL(hipdnnConvolutionForward(
        cudnnHandle_,
        &alpha,
        xDesc,
        x->data(),
        kernelDesc_,
        kernels->data(),
        convDesc_,
        HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
        nullptr,
        0,
        &beta,
        yDesc,
        y->data()));
  CUDNN_CALL(hipdnnAddTensor(
        cudnnHandle_,
        &alpha,
        biasDesc_,
        bias->data(),
        &alpha,
        yDesc,
        y->data()));
  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

void ConvolutionWrapper::backward(
    Tensor x,
    Tensor xGrad,
    Tensor kernels,
    Tensor kernelGrad,
    Tensor biasGrad,
    Tensor yGrad) {
  hipSetDevice(xGrad->getDevice());

  hipdnnTensorDescriptor_t xDesc, yDesc;
  setCudnnTensor(xDesc, xGrad);
  setCudnnTensor(yDesc, yGrad);

  const float alpha = 1.0f;
  const float beta = 1.0f;

  CUDNN_CALL(hipdnnConvolutionBackwardData(
        cudnnHandle_,
        &alpha,
        kernelDesc_,
        kernels->data(),
        yDesc,
        yGrad->data(),
        convDesc_,
        HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
        nullptr,
        0,
        &beta,
        xDesc,
        xGrad->data()));

  CUDNN_CALL(hipdnnConvolutionBackwardFilter(
      cudnnHandle_,
      &alpha,
      xDesc,
      x->data(),
      yDesc,
      yGrad->data(),
      convDesc_,
      HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
      nullptr,
      0,
      &beta,
      kernelDesc_,
      kernelGrad->data()));

  CUDNN_CALL(hipdnnConvolutionBackwardBias(
        cudnnHandle_,
        &alpha,
        yDesc,
        yGrad->data(),
        &beta,
        biasDesc_,
        biasGrad->data()));

  hipdnnDestroyTensorDescriptor(xDesc);
  hipdnnDestroyTensorDescriptor(yDesc);
}

ConvolutionWrapper::~ConvolutionWrapper() {
  hipdnnDestroyConvolutionDescriptor(convDesc_);
  hipdnnDestroyFilterDescriptor(kernelDesc_);
  hipdnnDestroyTensorDescriptor(biasDesc_);
}

void ConvolutionWrapper::setConvDescriptor(
    int hPad,
    int wPad,
    int hStride,
    int wStride) {
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&convDesc_));

  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        convDesc_,
        hPad,
        wPad,
        hStride,
        wStride,
        1,
        1,  // upscales
#if CUDNN_MAJOR > 5
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT));
#else
        HIPDNN_CROSS_CORRELATION));
#endif

}

void ConvolutionWrapper::setKernelDescriptor(const Shape& shape) {
  ABORT_IF(shape.size() != 4,
            "CUDN requires tensors 4D. Provided {}", shape.toString());
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&kernelDesc_));

  int layerIn = shape[0];
  int layerOut = shape[1];
  int kernelH = shape[2];
  int kernelW = shape[3];

  CUDNN_CALL(hipdnnSetFilter4dDescriptor(
        kernelDesc_,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        layerOut,
        layerIn,
        kernelH,
        kernelW));
}

}

#endif
