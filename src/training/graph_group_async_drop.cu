#include "hip/hip_runtime.h"
#include "training/graph_group_async.h"
#include "training/graph_group_async_drop.h"

#include "functional/functional.h"
#include "kernels/tensor_operators.h"
#include "training/dropper.h"
#include "training/sparse_tensor.h"

namespace marian {

Tensor AsyncGraphGroupDrop::newTensor(int size, int device) {
  Tensor t;
  Ptr<TensorAllocator> allocator_ = New<TensorAllocator>(device);
  allocator_->reserveExact(size * sizeof(float));
  allocator_->allocate(t, {1, size});
  allocators.push_back(allocator_);

  return t;
}

void AsyncGraphGroupDrop::fetchParams(Tensor oldParams,
                                      const std::vector<Tensor>& params,
                                      int device_id) {
  using namespace functional;
  // @TODO read guard on parameters
  int pos = 0;

  std::vector<std::thread> threads;
  for(int i = 0; i < devices_.size(); i++) {
    threads.emplace_back(std::thread(
        [&](int idx, int pos) {
          // individual mutex per-shard
          std::lock_guard<std::mutex> guard(shardSync_[idx]);

          // normal fetch
          if(fetchStep_[device_id] < FETCH_WARMUP
             || &params == &paramsAvg_) {  // Do not use sparse fetch when
                                           // fetching from paramsAvg
            oldParams->subtensor(pos, params[idx]->size())
                ->copyFrom(params[idx]);
            paramsLocal_[device_id][idx]->copyFrom(params[idx]);
            return;
          }

          // sparse fetch
          // get delta : params latest version - current param (locally)
          Element(_1 = _2 - _3,
                  paramsDelta_[idx],
                  params[idx],
                  paramsLocal_[device_id][idx]);

          // update current local param
          paramsLocal_[device_id][idx]->copyFrom(params[idx]);

          // get sparse delta
          fetchDropper[device_id][idx]->dropGraph(
              paramsDelta_[idx], fetchSparseGradient_[idx], droping_rate);

          // move sparse delta
          fetchShardedSparseGradient_[device_id][idx]->copyFrom(
              fetchSparseGradient_[idx]);

          fetchShardedSparseGradient_[device_id][idx]->scatterAdd(
              oldParams->subtensor(pos, params[idx]->size()));
        },
        i,
        pos));

    pos += shardSize_;
  }
  for(auto&& t : threads) {
    t.join();
  }
  fetchStep_[device_id]++;
}

void AsyncGraphGroupDrop::pushGradients(Tensor newGrads,
                                        size_t batch_words,
                                        int device_id) {
  if(pushStep_[device_id]++ < PUSH_WARMUP) {
    AsyncGraphGroup::pushGradients(newGrads, batch_words, device_id);
    return;
  }

  // get the sparse gradient
  pushDropper_[device_id]->dropGraph(
      newGrads, pushSparseGradient_[device_id], droping_rate);

  SparseTensor newSparseGrads = pushSparseGradient_[device_id];
  // add instead of copy?
  std::vector<std::thread> threads;
  int pos = 0;
  for(int idx = 0; idx < devices_.size(); idx++) {
    threads.emplace_back(std::thread(
        [=](int idx, int pos) {
          // individual mutex per-shard
          std::lock_guard<std::mutex> guard(shardSync_[idx]);

          // split to shard
          SparseTensor subGrad
              = newSparseGrads->subtensor(pos, grads_[idx]->size(), idx);

          // send the sharded sparse tensor
          pushShardedSparseGradient_[idx]->copyFrom(subGrad);

          // convert back to dense, store it in grads_[idx]
          pushShardedSparseGradient_[idx]->toDense(grads_[idx], -pos);

          if(scaleLearningRate_) {
            shardOpt_[idx]->update(
                params_[idx], grads_[idx], batch_words / avgBatchWords_);
          } else {
            shardOpt_[idx]->update(params_[idx], grads_[idx]);
          }

          if(movingAvg_)
            AsyncGraphGroup::updateMovingAverage(
                paramsAvg_[idx], params_[idx], scheduler_->numberOfBatches());

        },
        idx,
        pos));

    pos += shardSize_;
  }
  for(auto&& t : threads)
    t.join();
}

void AsyncGraphGroupDrop::init(Ptr<data::Batch> batch) {
  AsyncGraphGroup::init(batch);
  // extra inits for gradient dropping
  if(drop_first) {
    int totalSize = graphs_[0]->params()->vals()->size();
    int sparseCap = totalSize * 1.2 * (1.0 - 0.99);
    int shardSize = ceil(totalSize / devices_.size());

    for(int i = 0; i < devices_.size(); i++)
      paramsLocal_.push_back(std::vector<Tensor>());

    for(int i = 0; i < devices_.size(); i++) {
      // warm-up counter
      fetchStep_.push_back(0);
      pushStep_.push_back(0);

      int device = devices_[i];
      // temporary tensor to compute parameter delta before fetching
      paramsDelta_.push_back(newTensor(shardSize, device));

      // tensors to store local params history
      for(int h_id = 0; h_id < devices_.size(); h_id++) {
        Tensor tmp = newTensor(params_[i]->size(), device);
        tmp->copyFrom(params_[i]);
        paramsLocal_[h_id].push_back(tmp);
      }

      // individual Gradient dropper per-device
      pushDropper_.push_back(GradientDrop(new GradientDropBase()));

      // N-dropper for fetch
      std::vector<GradientDrop> tmpDropper;
      for(int i = 0; i < devices_.size(); i++)
        tmpDropper.push_back(GradientDrop(new GradientDropBase()));
      fetchDropper.push_back(tmpDropper);

      // sparsetensor to store sparsified gradients per-device
      pushSparseGradient_.push_back(
          SparseTensor(new SparseTensorBase(sparseCap, device)));

      pushShardedSparseGradient_.push_back(
          SparseTensor(new SparseTensorBase(sparseCap, device)));
      fetchSparseGradient_.push_back(SparseTensor(
          new SparseTensorBase(sparseCap / devices_.size(), device)));

      std::vector<SparseTensor> tmp;
      for(int i = 0; i < devices_.size(); i++)
        tmp.push_back(SparseTensor(
            new SparseTensorBase(sparseCap / devices_.size(), device)));
      fetchShardedSparseGradient_.push_back(tmp);
    }

    drop_first = false;
  }
}
}
