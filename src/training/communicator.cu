#include "hip/hip_runtime.h"
// @TODO: rename to communicator_nccl.h
// Note: This must only be included if defined(CUDA_FOUND) && defined(USE_NCCL)
// clang-format off
#include "training/communicator.h"
//#include "functional/functional.h"
//#include "tensors/tensor_operators.h"
// clang-format on
 
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "tensors/gpu/cuda_helpers.h"

namespace marian {

class NCCLCommunicator : public ICommunicator {
private:
  std::vector<ncclComm_t> comms_;     // [device index]
  std::vector<hipStream_t> streams_; // [device index]
  std::vector<int> devices_;          // [device index]
  Ptr<IMPIWrapper> mpi_; // non-null if multi-node

  static void groupStart() { NCCLCHECK(ncclGroupStart()); } // helpers to make sure we check the error
  static void groupEnd()   { NCCLCHECK(ncclGroupEnd());   }

  void synchronizeAll() {
    for(int i = 0; i < graphs_.size(); ++i) {
      CUDA_CHECK(hipSetDevice(devices_[i]));
      CUDA_CHECK(hipStreamSynchronize(streams_[i]));
    }
  }

  size_t myRank(size_t localDeviceIndex) const { // map local device index to a global rank
    if (mpi_)
      return mpi_->myRank() * devices_.size() + localDeviceIndex;
    else
      return localDeviceIndex;
  }

  size_t numWorkers() const { // total number of devices across all workers
    if (mpi_)
      return mpi_->commWorldSize() * devices_.size();
    else
      return devices_.size();
  }

  size_t dataSize() const { // total number of floats that comprise the concatenated parameter and gradient vector
    return graphs_[0]->params()->vals()->size();
  }

  // determine the (max) shard size
  // All shards except the last one have this size.
  // Presently, even all shards must have identical size, due to a limitation in NCCL we have not yet worked around.
  size_t shardSize() const {
    size_t numShards = numWorkers();
    size_t size = (dataSize() + numShards - 1) / numShards;
#if 1 // for now, all shards must have the same size, since NCCL does not allow a sub-slice for the last shard
    ABORT_IF(size * numShards != dataSize(), "presently, all shards must have the same size");
#endif
    return size;
  }

  // determine the index range (begin, end) of a shard
  std::pair<size_t, size_t> shardRange(size_t localDeviceIndex) const {
    size_t size = shardSize();
    size_t rank = myRank(localDeviceIndex);
    size_t begin = rank * size;
    size_t end = begin + size;
    end = std::min(end, dataSize()); // clip last shard. Note: Presently this never happens, since shardSize() enforces uniform shard size.
    return{ begin, end };
  }

public:
  // a NCCLCommunicator is bound to a set of graphs, one per GPU device
  // If MPI is used, then each worker has an instance of this class for its specific
  // set of GPU devices, which are communicating with each other.
  NCCLCommunicator(const std::vector<Ptr<ExpressionGraph>>& graphs, Ptr<IMPIWrapper> mpi)
      : ICommunicator(graphs),
        comms_(graphs.size()),
        streams_(graphs.size()),
        devices_(graphs.size()),
        mpi_(mpi) {
    if (mpi_)
      LOG(info, "[comm] Using NCCL library and MPI for GPU communication");
    else
      LOG(info, "[comm] Using NCCL library for GPU communication");

    for(int i = 0; i < graphs_.size(); ++i) {
      auto device = graphs_[i]->getBackend()->getDeviceId();

      ABORT_IF(device.type != DeviceType::gpu,
               "NCCL communicator can only be used with GPUs");

      devices_[i] = device.no;
      CUDA_CHECK(hipSetDevice(devices_[i]));
      CUDA_CHECK(hipStreamCreate(&streams_[i]));
    }

    // when using MPI, the setup is a laborious
    // cf. https://docs.nvidia.com/deeplearning/sdk/nccl-developer-guide/index.html#multidevprothrd
    if (mpi_) {
      // generate NCCL unique ID at one process and broadcast to all
      ncclUniqueId uniqueId = { 0 };
      LOG(info, "[{}] before ncclGetUniqueId", mpi_->to_string());
      if (mpi->myRank() == 0)
        NCCLCHECK(ncclGetUniqueId(&uniqueId));
      LOG(info, "[{}] before bcast", mpi_->to_string());
      //LOG(info, "before bcast: unique id = {}", std::string(uniqueId.internal, NCCL_UNIQUE_ID_BYTES));
      static_assert(sizeof(uniqueId) == NCCL_UNIQUE_ID_BYTES, "wrong NCCL_UNIQUE_ID_BYTES??"); // (this value is used in NVidia examples)
      mpi_->bCast(&uniqueId, sizeof(uniqueId), MPI_BYTE, 0);
      LOG(info, "[{}] after bcast", mpi_->to_string());
      //LOG(info, "unique id = {}", std::string(uniqueId.internal, NCCL_UNIQUE_ID_BYTES));

      // if more than one device then initialize NCCL with group API
      //if (devices_.size() > 1) {
        groupStart();
        for (int localDeviceIndex = 0; localDeviceIndex < devices_.size(); localDeviceIndex++) {
          CUDA_CHECK(hipSetDevice(devices_[localDeviceIndex]));
          LOG(info, "[{}] ncclCommInitRank {} out of {}, GPU[{}]", mpi_->to_string(), myRank(localDeviceIndex), numWorkers(), localDeviceIndex);
          NCCLCHECK(ncclCommInitRank(&comms_[localDeviceIndex], numWorkers(), uniqueId, myRank(localDeviceIndex)));
          LOG(info, "[{}] done ncclCommInitRank {} out of {}, GPU[{}]", mpi_->to_string(), myRank(localDeviceIndex), numWorkers(), localDeviceIndex);
        }
        groupEnd();
        LOG(info, "[{}] group done constructing NCCLCommunicator", mpi_->to_string());
      //}
      //// one device: no group API
      //else {
      //  CUDA_CHECK(hipSetDevice(devices_[0]));
      //  LOG(info, "[mpi rank {} of {}] ncclCommInitRank", mpi_->myRank(), mpi_->commWorldSize());
      //  NCCLCHECK(ncclCommInitRank(&comms_[0], mpi_->commWorldSize(), uniqueId, mpi_->myRank()));
      //  LOG(info, "[mpi rank {}] done constructing NCCLCommunicator", mpi_->myRank());
      //}
    }
    // without MPI, we have a handy convenience version to initialize
    // @TODO: We should be able to just use the code above as well.
    else {
      LOG(info, "ncclCommInitAll");
      NCCLCHECK(ncclCommInitAll(comms_.data(), devices_.size(), devices_.data()));
      LOG(info, "done ncclCommInitAll");
      LOG(info, "done constructing NCCLCommunicator");
    }
  }

  ~NCCLCommunicator() override {
    for(int i = 0; i < devices_.size(); ++i) {
      hipSetDevice(devices_[i]);
      hipStreamDestroy(streams_[i]);
      ncclCommDestroy(comms_[i]);
    }
  }

  void foreach(const std::function<void(size_t, size_t /*shardBegin*/, size_t /*shardEnd*/)>& func, bool parallel= true) const override {
    parallel &= graphs_.size() > 1;
      
    std::vector<std::thread> group;
    // iterate over all shards on this worker
    size_t begin, end;
    for(size_t i = 0; i < graphs_.size(); ++i) {
      std::tie
      (begin, end) = shardRange(i);
      //std::cerr << "[" << mpi_->to_string() << "] foreach " << begin << " " << end << std::endl;
      size_t size = end-begin;

      if (parallel)
        group.emplace_back(func, i, begin, end);
      else
        func(i, begin, end);
    }
    for(auto& t : group) // (note: group is empty is not parallel)
      t.join();
  }

  void scatterReduce() override {
    size_t begin, end;
    groupStart();
    for(int i = 0; i < graphs_.size(); ++i) {
      std::tie
      (begin, end) = shardRange(i);
      //std::cerr << "[" << mpi_->to_string() << "] scatterReduce " << begin << " " << end << std::endl;

      auto grads = graphs_[i]->params()->grads();
      const auto* sendbuf = grads->data();
      auto*       recvbuf = grads->subtensor(begin, end-begin)->data();
      size_t      bufsize = shardSize();

      NCCLCHECK(ncclReduceScatter(sendbuf, recvbuf, bufsize, ncclFloat, ncclSum, comms_[i], streams_[i]));
    }
    groupEnd();
    //std::cerr << "scatterReduce submitted" << std::endl;
    synchronizeAll();
    //std::cerr << "scatterReduce completed" << std::endl;
  }

  void allGather() override {
    size_t begin, end;
    groupStart();
    for(int i = 0; i < graphs_.size(); ++i) {
      std::tie
      (begin, end) = shardRange(i);
      //std::cerr << "[" << mpi_->to_string() << "] allGather " << begin << " " << end << std::endl;

      auto vals = graphs_[i]->params()->vals();
      const auto* sendbuf = vals->subtensor(begin, end-begin)->data();
      void*       recvbuf = vals->data();
      size_t      bufsize = shardSize();

      NCCLCHECK(ncclAllGather(sendbuf, recvbuf, bufsize, ncclFloat, comms_[i], streams_[i]));
    }
    groupEnd();
    synchronizeAll();
  }

  // swap paramShards worker[0].device[0] with a sharded set
  // This is used for the smoothed parameters, and also for persisting optimizer state.
  void swapParams(const std::vector<Tensor>& paramShards) override {
    ABORT_IF(mpi_ != nullptr, "swapParams() support for MPI is not yet implemented");
    // Update all graphs with parameter shard

    auto gather = [this, paramShards](size_t idx, size_t begin, size_t end) {
      // copy parameter shard to each graph, apart from last graph
      for(int i = 0; i < graphs_.size() - 1; ++i) {
        auto subParam
            = graphs_[i]->params()->vals()->subtensor(begin, end-begin);
        subParam->copyFrom(paramShards[idx]);
      }

      // back-up shard from last graph
      auto subParamLast
          = graphs_.back()->params()->vals()->subtensor(begin, end-begin);
      paramShards[idx]->copyFrom(subParamLast);

      auto subParamFirst
          = graphs_.front()->params()->vals()->subtensor(begin, end-begin);
      subParamLast->copyFrom(subParamFirst);
    };

    // execute for each shard
    foreach(gather);
  }

#if 0
  void pushParams(std::vector<Tensor>& paramShards) override {
    ABORT_IF(mpi_ != nullptr, "pushParams() support for MPI is not yet implemented");
    // Copy paramter shard from i-th graph to shard paramShards[i].
    // Graphs and shards with the same index live on the same device.

    auto copy = [this, paramShards](size_t idx, size_t begin, size_t end) {
      // copy parameter shard to each graph
      auto subParam
          = graphs_[idx]->params()->vals()->subtensor(begin, paramShards[idx]->size());
      paramShards[idx]->copyFrom(subParam);
    };

    foreach(copy);
  }

  void pullParams(const std::vector<Tensor>& paramShards) override {
    ABORT_IF(mpi_ != nullptr, "pullParams() support for MPI is not yet implemented");
    // Update all graphs with parameter shard

    auto gather = [this, paramShards](size_t idx, size_t begin, size_t end) {
      // copy parameter shard to each graph
      for(auto graph : graphs_) {
        auto subParam
            = graph->params()->vals()->subtensor(begin, paramShards[idx]->size());
        subParam->copyFrom(paramShards[idx]);
      }
    };
    foreach(gather);
  }
#endif

  // Doesn't work yet with NCCL
  // void pushParams(std::vector<Tensor>& params) {
  //   // Copy paramter shard from i-th graph to shard params[i].
  //   // Graphs and shards with the same index live on the same device.

  //   int pos = 0;
  //   for(int i = 0; i < graphs_.size(); ++i) {
  //     auto subParam = graphs_[i]->params()->vals()->subtensor(pos,
  //                                                             params[i]->size());
  //     groupStart();
  //     ncclBroadcast(subParam->data(),
  //                   params[i]->data(),
  //                   params[i]->size(),
  //                   ncclFloat,
  //                   0,
  //                   comms_[i],
  //                   streams_[i]);
  //     groupEnd();
  //     pos += params[i]->size();
  //   }
  //   synchronizeAll();
  // }

  // void pullParams(const std::vector<Tensor>& params) {
  //   // Update all graphs with parameter shard

  //   int totalSize = graphs_[0]->params()->vals()->size();
  //   int shardSize = ceil(totalSize / (float)graphs_.size());

  //   groupStart();
  //   for(int i = 0; i < graphs_.size(); ++i) {

  //     const void* sendbuff = (const void*)params[i]->data();
  //     void* recvbuff = (void*)graphs_[i]->params()->vals()->data();

  //     ncclAllGather(sendbuff,
  //                   recvbuff,
  //                   shardSize,
  //                   ncclFloat,
  //                   comms_[i],
  //                   streams_[i]);
  //   }
  //   groupEnd();

  //   synchronizeAll();
  // }
};

//Ptr<ICommunicator> newNCCLCommunicator(const std::vector<Ptr<ExpressionGraph>>& graphs, Ptr<IMPIWrapper> mpi) {
//  return New<NCCLCommunicator>(graphs, mpi);
//}

}  // namespace marian
