#include "hip/hip_runtime.h"
#include "optimizers.h"

#include "functional/functional.h"
#include "kernels/tensor_operators.h"

namespace marian {
void Sgd::updateImpl(Tensor params, Tensor grads) {
  using namespace functional;
  Element(_1 -= (multiplyFactor_ * eta_) * _2, params, grads);

  hipStreamSynchronize(0);
}

void Adagrad::updateImpl(Tensor params, Tensor grads) {
  if(!alloc_)
    alloc_ = New<TensorAllocator>(params->getBackend());

  if(!gt_) {
    int elements = params->size();
    alloc_->reserveExact(params->memory()->size());
    alloc_->allocate(gt_, {1, elements});
    gt_->set(0);
  }

  using namespace functional;

  Element(_1 += (_2 * _2), gt_, grads);

  Element(_1 -= ((multiplyFactor_ * eta_) / (sqrt(_2) + eps_)) * _3,
          params,
          gt_,
          grads);

  hipStreamSynchronize(0);
}

void Adagrad::resetStats() {
  if(gt_)
    gt_->set(0);
  hipStreamSynchronize(0);
}

void Adam::load(const std::string& name,
                std::vector<Ptr<OptimizerBase>> opts,
                std::vector<Ptr<Backend>> backends) {
  if(!boost::filesystem::exists(name))
    return;

  LOG(info, "Loading Adam parameters from {}", name);

  std::vector<float> vMt;
  std::vector<float> vVt;
  size_t totalSize = 0;

  auto numpy = cnpy::npz_load(name);
  for(auto it : numpy) {
    auto name = it.first;
    cnpy::NpyArray& np = it.second;

    // get the size of mt_ and vt_, they are the same
    totalSize = np.shape[1];

    // extract data into vectors
    if(name == "adam_mt") {
      vMt.resize(totalSize);
      std::copy((float*)np.data, (float*)np.data + totalSize, vMt.begin());
    }
    if(name == "adam_vt") {
      vVt.resize(totalSize);
      std::copy((float*)np.data, (float*)np.data + totalSize, vVt.begin());
    }
  }

  if(vMt.empty() || vVt.empty()) {
    LOG(info, "[warn] Adam parameters not found in .npz file");
    return;
  }

  size_t shardSize = ceil(totalSize / (float)backends.size());

  size_t id = 0;
  for(auto optBase : opts) {
    auto opt = std::dynamic_pointer_cast<Adam>(optBase);

    int size = std::min(shardSize, totalSize);
    totalSize -= size;

    if(!opt->mt_ || !opt->vt_) {
      if(!opt->alloc_)
        opt->alloc_ = New<TensorAllocator>(backends[id]);

      opt->alloc_->reserveExact(2 * sizeof(float) * size);
      opt->alloc_->allocate(opt->mt_, {1, size});
      opt->alloc_->allocate(opt->vt_, {1, size});
    }

    int shift = id * shardSize;
    std::vector<float> tmpMt(vMt.begin() + shift, vMt.begin() + shift + size);
    opt->mt_->set(tmpMt);
    std::vector<float> tmpVt(vVt.begin() + shift, vVt.begin() + shift + size);
    opt->vt_->set(tmpVt);

    id++;
  }
}

void Adam::save(const std::string& name,
                std::vector<Ptr<OptimizerBase>> opts,
                size_t totalSize) {
  LOG(info, "Saving Adam parameters to {}", name);

  std::vector<float> vMt;
  std::vector<float> vVt;

  for(auto optBase : opts) {
    auto opt = std::dynamic_pointer_cast<Adam>(optBase);

    std::vector<float> tmpMt;
    opt->mt_->get(tmpMt);
    vMt.insert(vMt.end(), tmpMt.begin(), tmpMt.end());

    std::vector<float> tmpVt;
    opt->vt_->get(tmpVt);
    vVt.insert(vVt.end(), tmpVt.begin(), tmpVt.end());
  }

  // truncate to the real size
  if(totalSize < vMt.size()) {
    vMt.resize(totalSize);
    vVt.resize(totalSize);
  }

  // the shape is the same for mt_ and vt_
  unsigned* shape = new unsigned[2];

  shape[0] = 1;
  shape[1] = vMt.size();

  cnpy::npz_save(name, "adam_mt", vMt.data(), shape, 2, "w");
  cnpy::npz_save(name, "adam_vt", vVt.data(), shape, 2, "a");

  delete[] shape;
}

void Adam::updateImpl(Tensor params, Tensor grads) {
  if(!alloc_)
    alloc_ = New<TensorAllocator>(params->getBackend());

  if(!mt_) {
    int elements = params->size();
    alloc_->reserveExact(2 * params->memory()->size());
    alloc_->allocate(mt_, {1, elements});
    mt_->set(0);

    alloc_->allocate(vt_, {1, elements});
    vt_->set(0);
  }

  t_++;
  float denom1 = 1 - std::pow(beta1_, t_);
  float denom2 = 1 - std::pow(beta2_, t_);

  using namespace functional;

  Element(_1 = (beta1_ * _1) + ((1 - beta1_) * _2), mt_, grads);
  Element(_1 = (beta2_ * _1) + ((1 - beta2_) * (_2 * _2)), vt_, grads);

  Element(_1 -= (multiplyFactor_ * eta_) * (_2 / denom1)
                / (sqrt(_3 / denom2) + eps_),
          params,
          mt_,
          vt_);

  hipStreamSynchronize(0);
}

void Adam::resetStats() {
  if(mt_)
    mt_->set(0);

  if(vt_)
    vt_->set(0);

  hipStreamSynchronize(0);
}

Ptr<OptimizerBase> Optimizer(Ptr<Config> options) {
  float lrate = options->get<double>("learn-rate");
  auto params = options->has("optimizer-params")
                    ? options->get<std::vector<float>>("optimizer-params")
                    : std::vector<float>({});

  Ptr<ClipperBase> clipper = nullptr;
  float clipNorm = options->get<double>("clip-norm");
  if(clipNorm > 0)
    clipper = Clipper<Norm>(clipNorm);

  auto opt = options->get<std::string>("optimizer");

  if(opt == "sgd") {
    return Optimizer<Sgd>(lrate, clipper, params);
  } else if(opt == "adagrad") {
    return Optimizer<Adagrad>(lrate, clipper, params);
  } else if(opt == "adam") {
    return Optimizer<Adam>(lrate, clipper, params);
  } else {
    ABORT("Unknown optimizer: {}", opt);
  }
}
}
